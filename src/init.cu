#include "hip/hip_runtime.h"
// Copyright (C) 2012,2013 Chi-kwan Chan
// Copyright (C) 2012,2013 Steward Observatory
//
// This file is part of GRay.
//
// GRay is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GRay is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
// License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GRay.  If not, see <http://www.gnu.org/licenses/>.

#include "gray.h"
#include "harm.h"
#include <cstdlib>
#include <para.h>
#include <ic.h>

static __global__ void kernel(State *s, const size_t n, const real t)
{
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n)
    s[i] = ic(i, n, t);
}

void init(Data &data)
{
  debug("init(*%p)\n", &data);

  const size_t n   = data;
  const size_t bsz = 64;
  const size_t gsz = (n - 1) / bsz + 1;

  State *s = data.device();
  kernel<<<gsz, bsz>>>(s, n, global::t);
  hipError_t err = hipDeviceSynchronize();
  data.deactivate();

  if(hipSuccess != err)
    error("init(): fail to launch kernel; %s\n", hipGetErrorString(err));
}

bool init_config(const char *arg)
{
  debug("init_config(""%s"")\n", arg);

  return config(arg[0], atof(arg + 2));
}
