#include "hip/hip_runtime.h"
// Copyright (C) 2012--2014 Chi-kwan Chan
// Copyright (C) 2012--2014 Steward Observatory
//
// This file is part of GRay.
//
// GRay is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// GRay is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
// License for more details.
//
// You should have received a copy of the GNU General Public License
// along with GRay.  If not, see <http://www.gnu.org/licenses/>.

#include "gray.h"

static __device__ __constant__ struct Counter {size_t *er;} count = {};

hipError_t Data::sync(size_t *p)
{
  debug("Data::sync(%p)\n", p);

  return hipMemcpyToSymbol(HIP_SYMBOL(count), &p, sizeof(size_t *));
}

static __device__ __constant__ Const c = {};

hipError_t Para::sync(Const *p)
{
  debug("Para::sync(%p)\n", p);

  return hipMemcpyToSymbol(HIP_SYMBOL(c), p, sizeof(Const));
}

#include <ic.h> // define device function ic()

static __global__ void kernel(State *s, const size_t n, const real t)
{
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n)
    s[i] = ic(i, n, t);
}

hipError_t Data::init(double t0)
{
  debug("Data::init(%g)\n", t0);

  kernel<<<gsz, bsz>>>(device(), n, t = t0);

  hipError_t err = hipDeviceSynchronize();
  if(hipSuccess == err)
    err = deactivate();
  return err;
}

#include <rhs.h>   // define device function rhs()
#include <getdt.h> // define device function getdt()

#define GET(s)  ((real *)&(s))[index]
#define EACH(s) for(int index = 0; index < NVAR; ++index) GET(s)
#  include <fixup.h>      // define device function fixup()
#  include "scheme/rk4.h" // define device function integrate()
#undef GET
#undef EACH

#ifdef PARTICLE_TIME
#  define GET_TIME (t = shared[threadIdx.x].PARTICLE_TIME)
#else
#  define GET_TIME t
#endif
#  include "scheme/driver.h" // define global kernel function driver()
#undef GET_TIME

hipError_t Data::evolve(double dt)
{
  debug("Data::evolve(%g)\n", dt);

  const double t0 = t, t1 = (t += dt);
  driver<<<gsz, bsz, bsz * sizeof(State)>>>(device(), n, t0, t1);

  hipError_t err = hipDeviceSynchronize();
  if(hipSuccess == err)
    err = deactivate();
  return err;
}
