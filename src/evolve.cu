#include "hip/hip_runtime.h"
// Copyright (C) 2012 Chi-kwan Chan
// Copyright (C) 2012 Steward Observatory
//
// This file is part of geode.
//
// Geode is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Geode is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
// License for more details.
//
// You should have received a copy of the GNU General Public License
// along with geode.  If not, see <http://www.gnu.org/licenses/>.

#include "geode.hpp"

static __global__ void kernel(State *s, size_t n, Real dt)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n) for(int h = 0; h < 100; ++h) {
    const Real x = s[i].x;
    const Real y = s[i].y;
    const Real z = s[i].z;
    const Real r = sqrt(x * x + y * y + z * z); // 6 FLOP
    const Real f = dt / (r * r * r);            // 3 FLOP

    s[i].x += dt * (s[i].u -= f * x); // 4 FLOP
    s[i].y += dt * (s[i].v -= f * y); // 4 FLOP
    s[i].z += dt * (s[i].w -= f * z); // 4 FLOP
  }
}

void evolve(void)
{
  const int bsz = 256;
  const int gsz = (global::n - 1) / bsz + 1;

  kernel<<<gsz, bsz>>>(global::s, global::n, 1.0e-3);
}
