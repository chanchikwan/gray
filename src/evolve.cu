#include "hip/hip_runtime.h"
// Copyright (C) 2012 Chi-kwan Chan
// Copyright (C) 2012 Steward Observatory
//
// This file is part of geode.
//
// Geode is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Geode is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
// License for more details.
//
// You should have received a copy of the GNU General Public License
// along with geode.  If not, see <http://www.gnu.org/licenses/>.

#include "geode.hpp"

#include <rhs.cu>

#ifndef BSZ
#define BSZ 256
#endif

#ifndef NLOOP
#define NLOOP 100
#endif

#ifndef FLOP
#define FLOP 0
#endif


static __global__ void kernel(State *s, size_t n, Real dt)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n) {
    const Real dt_2 = dt / 2;
    const Real dt_6 = dt / 6;

    State x = s[i];
    State y = x;

    for(int j = 0; j < NLOOP; ++j) {

      #define GET(s)  ((Real *)&(s))[index]
      #define EACH(s) for(int index = 0; index < NVAR; ++index) GET(s)

      const State k1 = rhs(y);
      #pragma unroll
      EACH(y) = GET(x) + dt_2 * GET(k1);

      const State k2 = rhs(y);
      #pragma unroll
      EACH(y) = GET(x) + dt_2 * GET(k2);

      const State k3 = rhs(y);
      #pragma unroll
      EACH(y) = GET(x) + dt   * GET(k3);

      const State k4 = rhs(y);
      #pragma unrol
      EACH(y) = GET(x) + dt_6 * (GET(k1) + 2 * (GET(k2) + GET(k3)) + GET(k4));

      #undef EACH
      #undef GET

      x = y;
    }

    s[i] = x;
  }
}

void evolve(void)
{
  using namespace global;

  hipEventRecord(c0, 0);
  {
    const int bsz = BSZ;
    const int gsz = (n - 1) / bsz + 1;

    kernel<<<gsz, bsz>>>(s, n, 1.0e-3);
  }
  hipEventRecord(c1, 0);
  hipEventSynchronize(c1);

  float ns;
  hipEventElapsedTime(&ns, c0, c1);
  ns /= NLOOP;

  std::cout
    << ns                                       << " ms/step, "
    << 1.0e-6 * n * (12 * NVAR + 4 * FLOP) / ns << " Gflops"
    << std::endl;
}
