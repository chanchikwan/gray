#include "hip/hip_runtime.h"
// Copyright (C) 2012 Chi-kwan Chan
// Copyright (C) 2012 Steward Observatory
//
// This file is part of geode.
//
// Geode is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Geode is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
// License for more details.
//
// You should have received a copy of the GNU General Public License
// along with geode.  If not, see <http://www.gnu.org/licenses/>.

#include "geode.hpp"

#define GET(s)  ((Real *)&(s))[index]
#define EACH(s) for(int index = 0; index < NVAR; ++index) GET(s)

#include <rhs.cu>
#include <rk4.cu>

static __global__ void kernel(State *state, size_t n, Real dt, size_t m)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n) {
    State s = state[i];
    for(int j = 0; j < m; ++j) s = rk4(s, dt);
    state[i] = s;
  }
}

void evolve(double dt, size_t nloop)
{
  using namespace global;

  hipEventRecord(c0, 0);
  {
    const int bsz = 256;
    const int gsz = (n - 1) / bsz + 1;

    kernel<<<gsz, bsz>>>(s, n, dt / nloop, nloop);
  }
  hipEventRecord(c1, 0);

  float ms;
  hipEventSynchronize(c1);
  hipEventElapsedTime(&ms, c0, c1);
  ms /= nloop;

  std::cout
    << ms                     << " ms/step, "
    << 1e-6 * flop() * n / ms << " Gflops"
    << std::endl;
}
