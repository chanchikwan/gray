#include "hip/hip_runtime.h"
// Copyright (C) 2012 Chi-kwan Chan
// Copyright (C) 2012 Steward Observatory
//
// This file is part of geode.
//
// Geode is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Geode is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
// License for more details.
//
// You should have received a copy of the GNU General Public License
// along with geode.  If not, see <http://www.gnu.org/licenses/>.

#include "geode.hpp"
#include <iomanip>

#define GET(s)  ((Real *)&(s))[index]
#define EACH(s) for(int index = 0; index < NVAR; ++index) GET(s)

#include <rhs.cu>
#include <rk4.cu>

static __global__ void kernel(State *state, size_t n, Real t, Real dt, size_t m)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n) {
    State s = state[i];

    for(int j = 0; j < m; ++j) {
      t += dt;

      // Only evolve the photon if it is outside the event horizon
      if(s.r > 1 + sqrt(1 - A_SPIN * A_SPIN)) s = scheme(s, t, dt);
    }

    state[i] = s;
  }
}

void evolve(double dt, size_t nloop)
{
  using namespace global;

  hipEventRecord(c0, 0);
  {
    const int bsz = 256;
    const int gsz = (n - 1) / bsz + 1;

    kernel<<<gsz, bsz>>>(s, n, t, dt / nloop, nloop);
    t += dt;
  }
  hipEventRecord(c1, 0);

  float ms;
  hipEventSynchronize(c1);
  hipEventElapsedTime(&ms, c0, c1);
  ms /= nloop;

  using namespace std;

  cout
    << fixed << setprecision(2)
    << "t = " << setw(6) << t << ", "
    << ms                     << " ms/step, "
    << 1e-6 * flop() * n / ms << " Gflops"
    << std::endl;
}
